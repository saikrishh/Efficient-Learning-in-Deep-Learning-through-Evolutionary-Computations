#include "hip/hip_runtime.h"
#include <math.h>
#include "config.h"
__device__ float sigmf( float val)
{
     return 1/(1+exp(-val));
    
}

__device__ void softmax(float *out,int l)
{
double s=0;
for( int i=0;i<l;i++)
    s=s+exp(out[i]); 
for( int i=0;i<l;i++)
{
   out[i]=expf(out[i])/s; 

if(out[i]!=out[i])
out[i]=1;
}

}


__device__ void computeNodeOut(float *out,int number,float *w,const float* Inputs)
{
int tx=threadIdx.x;
float sum=0;
if(number<InputCount)
{
out[number]=Inputs[tx*InputCount+number];
}
else
{
for(int i=0;i<number;i++)
sum=sum+w[i*n+number-i*(i+1)/2]*out[i];

sum=sum+w[number*n+number-number*(number+1)/2];
if(number>=n-TargetCount)
out[number]=sum;
  else
out[number]=sigmf(sum);
                
  }


}




__global__ void cost( float *costMatrix,const float* pop,const float* Inputs,const float* Targets)
{
    int bx = blockIdx.x;
    int tx = threadIdx.x;
float w[genome_length];
float NodeOut[n];
float out[TargetCount];

if(tx==2)
{
for(int i=0;i<genome_length;i++)
    w[i]=pop[bx*genome_length+i];


for(int i=0;i<n;i++)
{
computeNodeOut(NodeOut,i,w,Inputs);


}



float cost=0;
for(int i=0;i<TargetCount;i++)
out[TargetCount-i-1]=NodeOut[n-i-1];

softmax(out,TargetCount);

for(int i=0;i<TargetCount;i++)
{
if(out[i]<0.00000000001)
{
cost=cost-Targets[tx*TargetCount+i]*(-241.64);
}
else
{
cost=cost-Targets[tx*TargetCount+i]*(log2f(out[i]));
}
}

atomicAdd(&costMatrix[bx],cost);
}


}


