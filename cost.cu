#include "hip/hip_runtime.h"
#include <math.h>
#include "config.h"

__device__ float sigmf( float val)
{
     return 1/(1+exp(-val));
    
}


__device__ void computeNodeOut(float *out,int number,float *w,const float* Inputs)
{
int tx=threadIdx.x;
float sum=0;
if(number<InputCount)
{
out[number]=Inputs[tx*InputCount+number];
}
else
{
for(int i=0;i<number;i++)
sum=sum+w[i*n+number-i*(i+1)/2]*out[i];

sum=sum+w[number*n+number-number*(number+1)/2];
if(number>=n-TargetCount)
out[number]=sum;
  else
out[number]=sigmf(sum);
                
  }


}




__global__ void cost( float *costMatrix,const float* pop,const float* Inputs,const float* Targets)
{
    int bx = blockIdx.x;
    int tx = threadIdx.x;
float w[genome_length];
float NodeOut[n];


for(int i=0;i<genome_length;i++)
    w[i]=pop[bx*genome_length+i];


for(int i=0;i<n;i++)
computeNodeOut(NodeOut,i,w,Inputs);


float cost=0;
for(int i=0;i<TargetCount;i++)
cost=cost+pow(NodeOut[n-i-1]-Targets[tx*TargetCount+TargetCount-i-1],2);

atomicAdd(&costMatrix[bx],cost);



}


